#include "hip/hip_runtime.h"
#ifndef _2DCONVOLUTION_KERNEL_H_
#define _2DCONVOLUTION_KERNEL_H_

#include <stdio.h>
#include "2Dconvolution.h"

// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernel(Matrix N, Matrix P)
{
  __shared__ float N_s[BLOCK_SIZE][BLOCK_SIZE];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row_o = blockIdx.y * TILE_SIZE + ty;
  int col_o = blockIdx.x * TILE_SIZE + tx;

  int row_i = row_o - (KERNEL_SIZE/2);
  int col_i = col_o - (KERNEL_SIZE/2);

  float P_val = 0.0f;

  if((row_i >= 0) && (row_i < N.height) && (col_i >= 0) && (col_i < N.width)){
    N_s[ty][tx] = N.elements[row_i * N.width + col_i];
  }
  else{
    N_s[ty][tx] = 0.0;
  }
  __syncthreads();
    
  if(ty < TILE_SIZE && tx < TILE_SIZE ){
    for (int i = 0; i < KERNEL_SIZE; i++){
      for (int j = 0; j < KERNEL_SIZE; j++){
	P_val += Mc[i * KERNEL_SIZE + j] * N_s[i + ty][j + tx];
      }
    }
    __syncthreads();
  
    if(row_o < P.height && col_o < P.width){
      P.elements[row_o * P.width + col_o] = P_val;
    }
  }
}

#endif // #ifndef _2DCONVOLUTION_KERNEL_H_
